#include "hip/hip_runtime.h"
#include "linear-solver.h"
#include "contrib/utilities.h"
#include <iostream>
#include <string>
//TODO: In order to provide a fair service to all processes without exhausting GPU resources, we are creating an object pool to service instances of this class

// http://stackoverflow.com/questions/27827923/c-object-pool-that-provides-items-as-smart-pointers-that-are-returned-to-pool
using namespace std;
using namespace regression;
using namespace matrix;

static const char* LINEAR_SOLVER_ERROR_MESSAGES[] = {
	"@LinearSolver::LinearSolver(rows_, columns_). columns_ > rows_ ",
	"@LinearSolver::operator(matrix::Matrix & A, matrix::Matrix & B). Either A or B is empty.",
	"@LinearSolver::operator(matrix::Matrix & A, matrix::Matrix & B). Coefficient matrix's column count is greater solver's column count.",
	"@LinearSolver::operator(matrix::Matrix & A, matrix::Matrix & B). A.Rows() > rows || B.Rows() > rows || A.Rows() != B.Rows()"
};

__global__ void internal_copy_kernel(const double * __restrict d_in1, double * __restrict d_R, const double * __restrict d_C, double * __restrict d_B, const int M, const int N) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i < N) && (j < N)) {
        d_R[j * N + i] = d_in1[j * M + i];
        d_B[j * N + i] = d_C[j * M + i];
    }
}

__global__ void internal_identity_kernel(double * d_Q, const int M, const int N) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i < N) && (j < N) && (i == j))
    	d_Q[j * N + i] = 1.;
    else
    	d_Q[j * N + i] = 0.;
}

__global__ void internal_init_data(double * d_C, const int M /* Rows*/, const int N /* Columns */) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i < N) && (j < N))
    	d_C[j * N + i] = 1.0;
}


void LinearSolver::init(unsigned rows, unsigned columns){
	size_t Nrows = rows;
	size_t Ncols = columns;

	if(columns > rows)
		throw LinearSolverError((int)MORE_COLUMNS_THAN_ROWS, LINEAR_SOLVER_ERROR_MESSAGES[(int)MORE_COLUMNS_THAN_ROWS]);

    Grid = new dim3(iDivUp(columns, BLOCK_SIZE), iDivUp(columns, BLOCK_SIZE));
    Block = new dim3(BLOCK_SIZE, BLOCK_SIZE);

    // --- cuSOLVE input/output parameters/arrays
    gpuErrchk(hipMalloc(&devInfo, sizeof(int)));

    // --- CUDA solver initialization
    hipsolverDnCreate(&solver_handle);

    // --- CUBLAS initialization
    cublasSafeCall(hipblasCreate(&cublas_handle));

    // --- Setting the device matrix and moving the host matrix to the device
    gpuErrchk(hipMalloc((void**)&d_A,  Nrows * Ncols * sizeof(double)));

    // --- Creates d_Q d_TAU
    gpuErrchk(hipMalloc((void**)&d_TAU, min(Nrows, Ncols) * sizeof(double)));

    // --- CUDA QR initialization
    cusolveSafeCall(hipsolverDnDgeqrf_bufferSize(solver_handle, Nrows, Ncols, d_A, Nrows, &work_size));
    gpuErrchk(hipMalloc((void**)&work, work_size * sizeof(double)));

    // --- Creates d_Q
    gpuErrchk(hipMalloc((void**)&d_Q, Nrows * Nrows * sizeof(double)));

    // --- Creates d_D
    gpuErrchk(hipMalloc((void**)&d_D, Nrows  * Nrows  * sizeof(double)));

    // --- Creates d_R
    gpuErrchk(hipMalloc((void**)&d_R, Ncols * Ncols * sizeof(double)));

    // --- Creates d_B
    gpuErrchk(hipMalloc((void**)&d_B, Ncols * Ncols * sizeof(double)));

}

/** Creates the internal GPU & Host buffers.
 *  @throws LinearSystemException if there is not enough GPU RAM to create internal buffers.
 */
LinearSolver::LinearSolver(
	unsigned rows_ /**< [in] Strict positive integer. Must be >= columns_ */,
	unsigned columns_ /**< [in] Strict positive integer.  */
): rows(rows_), columns(columns_) {
	init(rows, columns);
	cout << "DEBUG: Allocator Constructor" << endl;
}


/** Copy constructor */
LinearSolver::LinearSolver (
		const LinearSolver& other /**< The source solver */
		):rows(other.rows), columns(other.columns) {
	init(rows, columns); // Dimension properties are the only ones copied.
	cout << "DEBUG: Copy Constructor" << endl;
}

/** Copy assignment operator */
LinearSolver & LinearSolver::operator = (const LinearSolver & other /**< The source matrix */){
	Destroy();
	init(rows, columns); // Dimension properties are the only ones copied.
	cout << "DEBUG: Copy assignment operator" << endl;
	return *this;
}

/** Move constructor */
LinearSolver::LinearSolver (LinearSolver&& other /**< The source solver */):rows(other.rows), columns(other.columns){
	init(rows, columns);
	CopyToThis(other);
	other.Clear();

	cout << "DEBUG: Move Constructor" << endl;
}

void LinearSolver::CopyToThis(LinearSolver & other ) {
	//Dynamically allocated
	this->d_A =other.d_A;  /**< Device buffer for coefficients */
	this->d_TAU=other.d_TAU; /**< Device buffer for τ */
	this->work=other.work; /**< Device intermediate buffer. */
	this->d_Q=other.d_Q; /**< Device buffer for CUDA QR execution. */
	this->d_D=other.d_D; /**< Device buffer for original/non-reduced right side matrix. */
	this->d_R=other.d_R; /**< Device buffer reduced right side matrix. */
	this->d_B=other.d_B; /**< Device buffer system's solution. */
	this->Grid=other.Grid;
	this->Block=other.Block;
	this->devInfo =other.devInfo; /**< device info pointer*/
	this->solver_handle =other.solver_handle; /**< solver handle */
	this->cublas_handle =other.cublas_handle;/**< cublas handle */

    //
	this->rows =other.rows; /** Row count (rows >=other.columns)*/
	this->columns =other.columns; /** Column count */
	this->work_size =other.work_size;
}

/** Move assignment operator */
LinearSolver& LinearSolver::operator = (LinearSolver&& other /**< The source matrix */)
{
	LinearSolver & ref = other;
    Destroy();
    CopyToThis(ref);
    other.Clear();

    cout << "DEBUG: Move assignment operator" << endl;
    return *this;
}

/** This () operator converts this class into a callable.
 *
 */
matrix::Matrix LinearSolver::operator () (
	matrix::Matrix & A /**< Coefficient Matrix */,
	matrix::Matrix & B /**< Right side matriz */
) {

	if(A.IsEmpty() || B.IsEmpty())
		throw LinearSolverError((int)EMPTY_OPERANDS, LINEAR_SOLVER_ERROR_MESSAGES[(int)EMPTY_OPERANDS]);

	if(A.Columns() > columns)
		throw LinearSolverError((int)COEFFICIENT_MATRIX_COLS_MISMATCH, LINEAR_SOLVER_ERROR_MESSAGES[(int)COEFFICIENT_MATRIX_COLS_MISMATCH]);

	if(A.Rows() > rows || B.Rows() > rows || A.Rows() != B.Rows())
		throw LinearSolverError((int)ROWS_MISMATCH, LINEAR_SOLVER_ERROR_MESSAGES[(int)ROWS_MISMATCH]);

    const int Nrows = (int)A.Rows();
    const int Ncols = (int)A.Columns();

	 // --- Setting the device matrix and moving the host matrix to the device
	const double * h_A = A.Buffer();
	gpuErrchk(hipMemcpy(d_A, h_A, Nrows * Ncols * sizeof(double), hipMemcpyHostToDevice));

   // --- CUDA GERF execution
	cusolveSafeCall(hipsolverDnDgeqrf(solver_handle, Nrows, Ncols, d_A, Nrows, d_TAU, work, work_size, devInfo));
	int devInfo_h = 0;
	gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));

	if (devInfo_h != 0)
		std::cout   << "Unsuccessful gerf execution\n\n";

    // --- At this point, the upper triangular part of A contains the elements of R. Showing this.
    gpuErrchk(hipMemcpy(( double *)h_A, d_A, Nrows * Ncols * sizeof(double), hipMemcpyDeviceToHost));

    // Initializes d_Q to identity
    internal_identity_kernel<<<*Grid, *Block>>>(d_Q, Nrows, Ncols);

    // --- CUDA QR execution
    cusolveSafeCall(hipsolverDnDormqr(solver_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_N, Nrows, Ncols, min(Nrows, Ncols), d_A, Nrows, d_TAU, d_Q, Nrows, work, work_size, devInfo));

    // --- Initializes d_D to ones.
    internal_init_data<<<*Grid, *Block>>>(d_D, Nrows, Nrows);

    // --- CUDA QR execution
    cusolveSafeCall(hipsolverDnDormqr(solver_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, Nrows, Ncols, min(Nrows, Ncols), d_A, Nrows, d_TAU, d_D, Nrows, work, work_size, devInfo));

    // --- Creates h_B
    Matrix cublasDstrmOutput(Ncols, Ncols);
    double *h_B = (double *)cublasDstrmOutput.Buffer();

    // --- Reducing the linear system size
    internal_copy_kernel<<<*Grid, *Block>>>(d_A, d_R, d_D, d_B, Nrows, Ncols);

    // --- Solving an upper triangular linear system
    const double alpha = 1.;
    cublasSafeCall(hipblasDtrsm(cublas_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, Ncols, Ncols,
                               &alpha, d_R, Ncols, d_B, Ncols));

    gpuErrchk(hipMemcpy(h_B, d_B, Ncols * Ncols * sizeof(double), hipMemcpyDeviceToHost));

	return cublasDstrmOutput;
}

void LinearSolver::Clear() {
	Grid = nullptr;
	Block = nullptr;
	devInfo = nullptr;
	solver_handle = nullptr;
	cublas_handle = nullptr;
	d_A = nullptr;
	d_TAU = nullptr;
	work = nullptr;
	d_Q = nullptr;
	d_D = nullptr;
	d_R = nullptr;
	d_B = nullptr;

	rows = 0;
	columns = 0;
    work_size = 0;
}

void LinearSolver::Destroy(){
	if(cublas_handle != nullptr){
		delete Grid;
		delete Block;
		hipFree(devInfo);
		hipsolverDnDestroy(solver_handle);
		hipblasDestroy(cublas_handle);
		hipFree(d_A);
		hipFree(d_TAU);
		hipFree(work);
		hipFree(d_Q);
		hipFree(d_D);
		hipFree(d_R);
		hipFree(d_B);
		Clear();
	}
}
LinearSolver::~LinearSolver() {
	Destroy();
}
